#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "cudarray/common.hpp"
#include "cudarray/random.hpp"


const char* curandGetErrorString(hiprandStatus_t error) {
  switch (error) {
  case HIPRAND_STATUS_SUCCESS:
    return "HIPRAND_STATUS_SUCCESS";
  case HIPRAND_STATUS_VERSION_MISMATCH:
    return "HIPRAND_STATUS_VERSION_MISMATCH";
  case HIPRAND_STATUS_NOT_INITIALIZED:
    return "HIPRAND_STATUS_NOT_INITIALIZED";
  case HIPRAND_STATUS_ALLOCATION_FAILED:
    return "HIPRAND_STATUS_ALLOCATION_FAILED";
  case HIPRAND_STATUS_TYPE_ERROR:
    return "HIPRAND_STATUS_TYPE_ERROR";
  case HIPRAND_STATUS_OUT_OF_RANGE:
    return "HIPRAND_STATUS_OUT_OF_RANGE";
  case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
    return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
  case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
    return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
  case HIPRAND_STATUS_LAUNCH_FAILURE:
    return "HIPRAND_STATUS_LAUNCH_FAILURE";
  case HIPRAND_STATUS_PREEXISTING_FAILURE:
    return "HIPRAND_STATUS_PREEXISTING_FAILURE";
  case HIPRAND_STATUS_INITIALIZATION_FAILED:
    return "HIPRAND_STATUS_INITIALIZATION_FAILED";
  case HIPRAND_STATUS_ARCH_MISMATCH:
    return "HIPRAND_STATUS_ARCH_MISMATCH";
  case HIPRAND_STATUS_INTERNAL_ERROR:
    return "HIPRAND_STATUS_INTERNAL_ERROR";
  }
  return "Unknown hiprand status";
}


namespace cudarray {

void seed(unsigned long long val) {
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(CURAND::generator(),
                                                  val));
}

template <>
void random_normal<float>(float *a, float mu, float sigma, unsigned int n) {
  CURAND_CHECK(hiprandGenerateNormal(CURAND::generator(), a, n, mu, sigma));
}


template<typename T>
__global__ void kernel_stretch(T *a, T alpha, T beta, unsigned int n) {
  CUDA_GRID_STRIDE_LOOP(idx, n) {
    a[idx] = alpha*a[idx] + beta;
  }
}


template <>
void random_uniform<float>(float *a, float low, float high, unsigned int n) {
  CURAND_CHECK(hiprandGenerateUniform(CURAND::generator(), a, n));
  if (high != 1.0 || low != 0.0) {
    float alpha = high - low;
    float beta = low;
    kernel_stretch<<<CUDA_BLOCKS(n), CUDA_NUM_THREADS>>>(a, alpha, beta, n);
  }
}

}
