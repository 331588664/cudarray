#include "hip/hip_runtime.h"
#include <climits>
#include <cfloat>
#include <hip/hip_math_constants.h>
#include "cudarray/common.hpp"
#include "cudarray/reduction.hpp"

// TODO: Parallel reductions! See
// http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

namespace cudarray {

#define REDUCE_OP(name, ident_f, ident_i, reduce_op, scale_op, select_op) \
template <typename Tb> \
struct name; \
template <> \
struct name<float> { \
  const static float identity = ident_f; \
  template <typename Ta, typename Tb> \
  __device__ inline static void reduce(volatile Ta a, volatile int idx, \
                                       volatile Tb &b, volatile int &b_idx) { \
    reduce_op; \
  } \
  template <typename Tb> \
  __device__ inline static void scale(volatile Tb &b, volatile float n) { \
    scale_op; \
  } \
  template <typename Ta, typename Tb> \
  __device__ inline static void select(volatile Tb &b, volatile Ta a, \
                                       volatile int idx) { \
    select_op; \
  } \
}; \
template <> \
struct name<int> { \
  const static int identity = ident_i; \
  template <typename Ta, typename Tb> \
  __device__ inline static void reduce(volatile Ta a, volatile int idx, \
                                       volatile Tb &b, volatile int &b_idx) { \
    reduce_op; \
  } \
  template <typename Tb> \
  __device__ inline static void scale(volatile Tb &b, volatile float n) { \
    scale_op; \
  } \
  template <typename Ta, typename Tb> \
  __device__ inline static void select(volatile Tb &b, volatile Ta a, \
                                       volatile int idx) { \
    select_op; \
  } \
};

REDUCE_OP(max_op, -FLT_MAX, INT_MIN, if (a > b) b = a, , b = a)
REDUCE_OP(mean_op, 0.0f, 0, b += a, b /= n, b = a)
REDUCE_OP(min_op, FLT_MAX, INT_MAX, if (a < b) b = a, , b = a)
REDUCE_OP(sum_op, 0.0f, 0, b += a, , b = a)
REDUCE_OP(argmax_op, -FLT_MAX, INT_MIN, if (a > b) {b = a; b_idx=idx;}, , b = idx)
REDUCE_OP(argmin_op, FLT_MAX, INT_MAX, if (a < b) {b = a; b_idx=idx;}, , b = idx)



template <typename Ta, typename Tb, typename Op>
__global__ void kernel_reduce(const Ta *a, unsigned int n, Tb *b) {
  CUDA_GRID_STRIDE_LOOP(idx, 1) {
    Ta a_ = Op::identity;
    int idx_ = 0;
    for (unsigned int i = 0; i < n; ++i) {
      Op::reduce(*a, i, a_, idx_);
      ++a;
    }
    Op::scale(a_, n);
    Op::select(*b, a_, idx_);
//    *b = b_;
  }
}


template <typename Ta, typename Tb, typename Op>
void reduce(const Ta *a, unsigned int n, Tb *b) {
  kernel_reduce<Ta, Tb, Op><<<CUDA_BLOCKS(1), CUDA_NUM_THREADS>>>(a, n, b);
}


template<typename T>
void reduce(ReduceOp op, const T *a, unsigned int n, T *b) {
  switch (op) {
    case MAX_OP:
      reduce<T, T, max_op<T> >(a, n, b);
      break;
    case MEAN_OP:
      reduce<T, T, mean_op<T> >(a, n, b);
      break;
    case MIN_OP:
      reduce<T, T, min_op<T> >(a, n, b);
      break;
    case SUM_OP:
      reduce<T, T, sum_op<T> >(a, n, b);
      break;
  }
}

template void reduce<float>(ReduceOp op, const float *a, unsigned int n,
                            float *b);
template void reduce<int>(ReduceOp op, const int *a, unsigned int n,
                            int *b);


template<typename T>
void reduce_to_int(ReduceToIntOp op, const T *a, unsigned int n, int *b) {
  switch (op) {
    case ARGMAX_OP:
      reduce<T, int, argmax_op<T> >(a, n, b);
      break;
    case ARGMIN_OP:
      reduce<T, int, argmin_op<T> >(a, n, b);
      break;
  }
}

template void reduce_to_int<float>(ReduceToIntOp op, const float *a,
                                   unsigned int n, int *b);
template void reduce_to_int<int>(ReduceToIntOp op, const int *a,
                                 unsigned int n, int *b);





template <typename Ta, typename Tb, typename Op, bool reduce_leading>
__global__ void kernel_reduce_mat(const Ta *a, unsigned int m, unsigned int n,
                                  Tb *b) {
  unsigned int n_threads;
  if (reduce_leading) {
    n_threads = n;
  } else {
    n_threads = m;
  }

  CUDA_GRID_STRIDE_LOOP(idx, n_threads) {
    if (reduce_leading) {
      a += idx;
      b += idx;
    } else {
      a += idx * n;
      b += idx;
    }

//    Tb b_ = Op::identity;
    Ta a_ = Op::identity;
    int idx_ = 0;
    if (reduce_leading) {
      for (unsigned int i = 0; i < m; ++i) {
        Op::reduce(*a, i, a_, idx_);
//        Op::reduce(*a, b_, i);
        a += n;
      }
    } else {
      for (unsigned int i = 0; i < n; ++i) {
        Op::reduce(*a, i, a_, idx_);
//        Op::reduce(*a, b_, i);
        ++a;
      }
    }

    if (reduce_leading) {
      Op::scale(a_, m);
//      Op::scale(b_, m);
    } else {
      Op::scale(a_, n);
//      Op::scale(b_, n);
    }
    Op::select(*b, a_, idx_);
//    *b = b_;
  }
}

template<typename Ta, typename Tb, typename Op>
void reduce_mat(const Ta *a, unsigned int m, unsigned int n,
                bool reduce_leading, Tb *b) {
  if (reduce_leading) {
    kernel_reduce_mat<Ta, Tb, Op, true><<<CUDA_BLOCKS(n), CUDA_NUM_THREADS>>>
        (a, m, n, b);
  } else {
    kernel_reduce_mat<Ta, Tb, Op, false><<<CUDA_BLOCKS(m), CUDA_NUM_THREADS>>>
        (a, m, n, b);
  }
}

template<typename T>
void reduce_mat(ReduceOp op, const T *a, unsigned int m, unsigned int n,
                bool reduce_leading, T *b) {
  switch (op) {
    case MAX_OP:
      reduce_mat<T, T, max_op<T> >(a, m, n, reduce_leading, b);
      break;
    case MEAN_OP:
      reduce_mat<T, T, mean_op<T> >(a, m, n, reduce_leading, b);
      break;
    case MIN_OP:
      reduce_mat<T, T, min_op<T> >(a, m, n, reduce_leading, b);
      break;
    case SUM_OP:
      reduce_mat<T, T, sum_op<T> >(a, m, n, reduce_leading, b);
      break;
  }
}

template void reduce_mat<float>(ReduceOp op, const float *a, unsigned int m,
                                unsigned int n, bool reduce_leading, float *b);
template void reduce_mat<int>(ReduceOp op, const int *a, unsigned int m,
                                unsigned int n, bool reduce_leading, int *b);


template<typename T>
void reduce_mat_to_int(ReduceToIntOp op, const T *a, unsigned int m,
                       unsigned int n, bool reduce_leading, int *b) {
  switch (op) {
    case ARGMAX_OP:
      reduce_mat<T, int, argmax_op<T> >(a, m, n, reduce_leading, b);
      break;
    case ARGMIN_OP:
      reduce_mat<T, int, argmin_op<T> >(a, m, n, reduce_leading, b);
      break;
  }
}

template void reduce_mat_to_int<float>(ReduceToIntOp op, const float *a,
    unsigned int m, unsigned int n, bool reduce_leading, int *b);
template void reduce_mat_to_int<int>(ReduceToIntOp op, const int *a,
    unsigned int m, unsigned int n, bool reduce_leading, int *b);

}
